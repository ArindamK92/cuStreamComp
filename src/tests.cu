#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>
#include "cuCompactor.cuh"
#include <chrono>
using namespace std;


struct int_predicate
{
	__host__ __device__
	bool operator()(const int x)
	{
		return x>0;
	}
};

void initiData(int *h_data, uint NELEMENTS,uint &goodElements){
	ushort stride = 4;
	for (int i = 0; i < NELEMENTS; ++i) {
		h_data[i] = i%stride;
		if(h_data[i])
			goodElements++;
	}
}


void printData(int *h_data, uint NELEMENTS){
	for (int i = 0; i < NELEMENTS; ++i) {
		cout<<h_data[i]<<" ";
	}
	cout<<endl;
}


void checkVector(int *h_data,uint NELEMENTS,uint NgoodElements){
	//printf("Checking: %i, %i",NELEMENTS,NgoodElements);
	int_predicate predicate;
	for(int i=0;i<NgoodElements;i++){
		assert(predicate(h_data[i]));
	}
	for(int i=NgoodElements;i<NELEMENTS;i++){
		assert(!predicate(h_data[i]));
	}
}



unsigned int NELEMENTS=0;
uint NgoodElements=0;
uint blockSize=8;


int main(){
	int *d_data, *d_output, *h_data;

	//data elements from 2^5 to 2^29
	for(int e=7;e<30;e++){
	//blocksize from 32 to 1024
		for(int b=5;b<=10;b++){

			NELEMENTS=1<<e;
			NgoodElements=0;
			blockSize=1<<b;
			size_t datasize=sizeof(int)*NELEMENTS;
			//host input/output data
			h_data = (int*) malloc(datasize);
			memset(h_data,0,datasize);
			//device input data
			hipMalloc(&d_data,datasize);
			//device output data
			hipMalloc(&d_output,datasize);

			hipMemset(d_output,0,datasize);
			initiData(h_data,NELEMENTS,NgoodElements);

			//printData(h_data,NELEMENTS);

			hipMemcpy(d_data,h_data,datasize,hipMemcpyHostToDevice);

			clock_t start = clock();
			cuCompactor::compact<int>(d_data,d_output,NELEMENTS,int_predicate(),blockSize);
			hipDeviceSynchronize();
			clock_t end = clock();
			unsigned long millis = (end - start) * 1000 / CLOCKS_PER_SEC;


			//copy back results to host
			hipMemcpy(h_data,d_output,datasize,hipMemcpyDeviceToHost);
			//printData(h_data,NELEMENTS);
			checkVector(h_data,NELEMENTS,NgoodElements);
			//device memory free
			hipFree(d_data);
			hipFree(d_output);
			//host free  memory
			free(h_data);
			printf("(%i,%i,%i)\n",NELEMENTS,blockSize,millis);
		}//for blocksize
	}//for elements
	printf("ALL TEST PASSED");

}
